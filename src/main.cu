//19/10/17
//ALOUI Driss
//DO Alexandre
//HPCA

//Source : https://www.cc.gatech.edu/~bader/papers/GPUMergePath-ICS2012.pdf

//Exemple de tests
//N: 5463 GRAIN: 10
//Time GPU : 3.856832 ms
//Time CPU : 6.504000 ms

//N: 100 GRAIN: 10
//Time GPU : 0.141312 ms
//Time CPU : 0.109536 ms

//---------------- INCLUDE ----------------
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>
#include <fstream>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <string>

//---------------- DEFINE ----------------
#define NB 1024
#define NTPB 1024
#define N NTPB*NB

#define TAILLE 13
#define GRAIN 1024

//---------------- FONCTIONS UTILITAIRES ----------------
//Fct test tri
bool is_sorted(int* array, int n) {
  for(int i=0; i<n-1; i++) {
    if(array[i]>array[i+1]) return false;
  }
  return true;
}

//Fct test egale
bool is_equal(int* array1, int* array2, int n) {
  for(int i=0; i<n-1; i++) {
    if(array1[i]!=array2[i]) return false;
  }
  return true;
}


//Fct Pgcd (pour la gestion du nombre de thread)
// Source : https://openclassrooms.com/forum/sujet/algorithme-de-calcul-de-pgcd-20803
int get_pgcd(int a, int b){

  int pgcd = 0;
  while(1){
    pgcd = a % b;
      if(pgcd == 0){
        pgcd = b;
        break;
      }
    a = b;
    b = pgcd;
  }
  return pgcd;
}

//---------------- version CPU ----------------

//Fct Merge CPU.
void MergeCPU(int *A, int *L, int leftCount, int *R, int rightCount) {
  int i = 0, j = 0, k = 0;
  while(i<leftCount && j<rightCount) {
    if(L[i] < R[j]) 
      A[k++] = L[i++];
    else A[k++] = R[j++];
  }
  while(i<leftCount) 
    A[k++] = L[i++];
  while(j<rightCount) 
    A[k++] = R[j++];
}

// Fct Merge and sort pour CPU
void mergeAndSortRecuCPU(int *A,int n) {
  int mid,i, *L, *R;
  if(n < 2) 
    return; // s'il y a moins de deux elements, on ne fait rien

  mid = n/2;

  // creation tableaux de gauche et de droite
  // de 0 à mid - 1 = gauche, il y a mid elements
  // de mid à n-1 = droite, il y a n-mid elements
  L = (int*)malloc(mid*sizeof(int)); 
  R = (int*)malloc((n-mid)*sizeof(int)); 

  for(i = 0;i<mid;i++) 
    L[i] = A[i]; 
  for(i = mid;i<n;i++) 
    R[i-mid] = A[i];

  mergeAndSortRecuCPU(L,mid);  // tri tableau de gauche
  mergeAndSortRecuCPU(R,n-mid);  // tri tableau de droite
  MergeCPU(A,L,mid,R,n-mid);  // fusion des tableaux
  free(L);
  free(R);
}



//---------------- version GPU ----------------

//Fct Merge GPU
__host__ __device__ void mergeGPU(int* A, int na, int aid, int* B, int nb, int bid, int* C, int cid, int T) {

  for(int i=0; i<T; i++) {
    if(aid<na && bid<nb) {
      if(A[aid] < B[bid]) {
        C[cid+i] = A[aid];
        aid++;
      }else {
        C[cid+i] = B[bid];
        bid++;
      }
    }else{ // Pour derniers indices (reste plus q un tableau)
      if(aid<na){
        C[cid+i] = A[aid];
        aid++;
      }else{
        C[cid+i] = B[bid];
        bid++;
      }
    }
  }
}

//Fct GPU partitionning GPU
__global__ void partitionningGPU(int* A, int na, int* B, int nb, int* C){
  int a, b, offset;
  int aid;
  int bid;

  int tid = blockIdx.x*blockDim.x+threadIdx.x;// identifiant de thread
  int index = tid*(na+nb)/(blockDim.x * gridDim.x);// index de debut dans C
  int a_top = (index>na)? na:index;
  int b_top = (index>na)? index-na:0;
  int a_bot = b_top;
  if(tid != 0) {
// binary search for diagonal intersectios
    while(true) {
      offset = (a_top - a_bot) / 2;
      a = a_top - offset;
      b = b_top + offset;

      if(A[a]>B[b-1]){
        if(A[a-1]<=B[b]){
          aid = a;
          bid = b;
          break; //point trouve !
        }else{
          a_top = a-1;
          b_top = b+1;
        }
      }else{
        a_bot = a+1;
      }
    }
  }else{
    aid = 0;
    bid = 0;
  }
// merge
printf("[%d] Call merge\n",tid);
  mergeGPU(A, na, aid, B, nb, bid, C, index, (na+nb)/(blockDim.x * gridDim.x));
}

// Fct Merge and sort pour GPU
void mergeAndSortRecuGPU(int T[], int i_debut, int i_fin)
{
  int n = i_fin - i_debut + 1;
/*
printf("\n ------ i_debut %d i_fin %d blockSize %d \n",i_debut,i_fin, blockSize);
 printf("\n T %d : \n",n);
   for(int i=0;i<n;i++){
  printf("%d\t",T[i]);
    }
  printf("\n");
*/
  if (n < GRAIN){

    int i_milieu = i_debut + (i_fin - i_debut) / 2;

    int na = 1 + i_milieu - i_debut;
    int nb = i_fin - i_milieu;
    
    mergeAndSortRecuGPU(T, i_debut, i_milieu);
    mergeAndSortRecuGPU(T, i_milieu+1, i_fin);
  
    int *A, *B, *C;
    hipMalloc(&A, na*sizeof(int));
    hipMalloc(&B, nb*sizeof(int));
    hipMalloc(&C, n*sizeof(int));

    if(!A || !B || !C ) {
      printf("[Error] memory alloc error\n");
      return;
    }

  //Cpu vers Gpu
    int error1 = hipMemcpy(A, T, na*sizeof(int), hipMemcpyHostToDevice);
    int error2 = hipMemcpy(B, T+na, nb*sizeof(int), hipMemcpyHostToDevice);
    if(error1)
      printf("[Error] error1 %d (Cpu vers Gpu)\n",error1);
    if(error2)
      printf("[Error] error2 %d (Cpu vers Gpu)\n",error2);

  //partitionning
printf("\nCall GPUpartitionning NB %d NTPB %d na %d nb %d\n",NB,get_pgcd(na, nb),na,nb);
    partitionningGPU<<<NB,NTPB>>>(A, na, B, nb, C);

  //Gpu vers cpu
    int error3 = hipMemcpy(T, C, n*sizeof(int), hipMemcpyDeviceToHost);
    if(error3)
      printf("[Error] error3 %d (Gpu vers cpu)\n",error3);

  //free
    free(T);
    hipFree(A);
    hipFree(B);
    hipFree(C);

  }else{
  //tri
    mergeAndSortRecuCPU(T, n);
  }
}

//---------------- FONCTIONS APP ----------------

// Fct intro
void intro(){
//INTRODUCTION Longue
  system("clear");
  printf("Dans le cadre\n");
  printf("\n");
  sleep(1);
  printf("N9-IPA PARALLELISME AVANCE HPCA\n");
  printf("Projet de fin de module\n");
  printf("\n");
  sleep(1);
  printf("2017 - 2018\n");
  sleep(2);

  system("clear");
  printf("Supervisé par\n");
  printf("\n");
  printf("Lokman ABBAS TURKI\n");
  sleep(2);

  system("clear");
  printf("ALOUI Driss\n");
  printf("DO Alexandre\n");
  printf("\n");
  sleep(1);
  printf("MAIN 5 Polytech Paris UPMC\n");
  sleep(2);

  system("clear");
  printf("Présentent\n");
  sleep(1);

  system("clear");
  printf("MergeAndSortAppInCuda\n");
  printf("\n");
  sleep(1);
  printf("Version finale\n");
  sleep(1);

  system("clear");
  char s;
  printf("Appuyer sur ENTREE pour continuer\n");
  s=getchar();
  putchar(s);
  system("clear");
}


// Fct intro
void introShort(){
//INTRODUCTION courte
  system("clear");
  printf("Dans le cadre\n");
  printf("\n");
  printf("N9-IPA PARALLELISME AVANCE HPCA\n");
  printf("Projet de fin de module\n");
  printf("\n");
  printf("2017 - 2018\n");
  sleep(2);

  system("clear");
  printf("ALOUI Driss\n");
  printf("DO Alexandre\n");
  printf("\n");
  sleep(2);

  system("clear");
  printf("Présentent\n");
  sleep(1);

  system("clear");
  printf("MergeAndSortAppInCuda\n");
  printf("\n");
  printf("Version finale\n");
  sleep(1);

  system("clear");
  char s;
  printf("Appuyer sur ENTREE pour continuer\n");
  s=getchar();
  putchar(s);
  system("clear");
}

//2 Test sur tableau random
void tabRandom(){
  int x;
  int Entree = 0;
  int Erreur = 0;
  while(Entree == 0){
    system("clear");
    std::cout<<"Test sur tableau random"<<std::endl;

    if(Erreur == 1){
      std::cout<<"[Erreur] Entrée non valide ! (entier positif)"<<std::endl;
      Erreur  =  0;
    }
    std::cout<<"Entrer la taille du tableau puis valider avec la touche ENTREE"<<std::endl;
    std::cin>>x;
    while (getchar() != '\n'); //vide le buffer de saisie
    if(x>0){
      Entree = 1;
      char s;
      printf("Vous avez entré %d\n", x);
      printf("Appuyer sur ENTREE pour continuer\n");
      s=getchar();
      putchar(s);
    }else{
      Erreur  =  1;
    }
  }

  int n = x;

// var pour timer
  hipEvent_t startCPU, stopCPU;
  hipEventCreate ( &startCPU );
  hipEventCreate ( &stopCPU );
  hipEvent_t startGPU, stopGPU;
  hipEventCreate ( &startGPU );
  hipEventCreate ( &stopGPU );

//Alloc Array
printf("Alloc Array\n");
  srand(time(NULL));
  int* T_cpu = (int*)malloc(n*sizeof(int));
  int* T_gpu = (int*)malloc(n*sizeof(int));
  int cpt=0;

//init Array
printf("\nInit Array\n");
  while (cpt<n){
    T_gpu[cpt]=(rand()%100);
    T_cpu[cpt] = T_gpu[cpt];
    cpt++;
  }
printf("\n");

//sort CPU
printf("\nCall sort CPU\n");
  hipEventRecord(startCPU);
  mergeAndSortRecuGPU(T_gpu, 0, n-1);
  hipEventRecord(stopCPU);
printf("\n");
//sort GPU
printf("\nCall sort GPU\n");
  hipEventRecord(startGPU);
  mergeAndSortRecuCPU(T_cpu, n);
  hipEventRecord(stopGPU);
printf("Call cudaDeviceSynchronize\n");
  hipDeviceSynchronize();

//test tri ok?
printf("\nTest tri\n");
  if(is_sorted(T_cpu, n)){
    printf("OK\n");
  //Fct test egale
printf("Test egale\n");
    if(is_equal(T_cpu, T_gpu, n))
      printf("OK\n");
    else
      printf("[error] T_gpu mal trie");
  }else{
    printf("[error] T_cpu mal trie");   
  }

//Time resuts
printf("\nTime results\n");
  float millisecondsGPU = 0;
  hipEventElapsedTime(&millisecondsGPU, startGPU, stopGPU);
  printf("\nTime GPU : %f ms\n",millisecondsGPU);
  float millisecondsCPU = 0;
  hipEventElapsedTime(&millisecondsCPU, startCPU, stopCPU);
  printf("Time CPU : %f ms\n",millisecondsCPU);

printf("\n");
  char s;
  printf("Appuyer sur ENTREE pour continuer\n");
  s=getchar();
  putchar(s);

//free
  free(T_cpu);
  free(T_gpu);

}

//3 Lecture à partir d'un fichier
void tabFile(){
  std::ifstream file;
  std::string x;
  int Entree = 0;
  int Erreur = 0;
  while(Entree == 0){
    system("clear");
    std::cout<<"Lecture à partir d'un fichier"<<std::endl;

    if(Erreur == 1){
      std::cout<<"[Erreur] Entrée non valide ! Le fichier doit se trouver dans le repertoire courant ex : exemple.txt et il doit utiliser des retours à la ligne en séparateur"<<std::endl;
      Erreur  =  0;
    }
    std::cout<<"Entrer le nom du fichier puis valider avec la touche ENTREE"<<std::endl;
    std::cin>>x;
    while (getchar() != '\n'); //vide le buffer de saisie
    x = "../data/" + x;
    file.open(x);
    if(file.is_open()){
      char s;
      std::cout<<"Vous avez entré "<<x<<std::endl;
      printf("Appuyer sur ENTREE pour continuer\n");
      s=getchar();
      putchar(s);
      Entree = 1;
    }else{
      Erreur  =  1;
    }
  }

//ouverture fichier

//lecture fichier
printf("Lecture fichier...\n");
  int num;
  std::vector<int> tab;
  while (file >> num) { tab.push_back(num); }

  int n = tab.size();
printf("%d elements lues\n",n);

// var pour timer
  hipEvent_t startCPU, stopCPU;
  hipEventCreate ( &startCPU );
  hipEventCreate ( &stopCPU );
  hipEvent_t startGPU, stopGPU;
  hipEventCreate ( &startGPU );
  hipEventCreate ( &stopGPU );

//Alloc Array
printf("Alloc Array\n");
  srand(time(NULL));
  int* T_cpu = (int*)malloc(n*sizeof(int));
  int* T_gpu = (int*)malloc(n*sizeof(int));
  int cpt=0;

//init Array
printf("\nInit Array\n");
  while (cpt<n){
    T_gpu[cpt]= tab[cpt];
    T_cpu[cpt] = tab[cpt];
    cpt++;
  }
printf("\n");

//sort CPU
printf("\nCall sort CPU\n");
  hipEventRecord(startCPU);
  mergeAndSortRecuGPU(T_gpu, 0, n-1);
  hipEventRecord(stopCPU);
printf("\n");
//sort GPU
printf("\nCall sort GPU\n");
  hipEventRecord(startGPU);
  mergeAndSortRecuCPU(T_cpu, n);
  hipEventRecord(stopGPU);
printf("Call cudaDeviceSynchronize\n");
  hipDeviceSynchronize();

//test tri ok?
printf("\nTest tri\n");
  if(is_sorted(T_cpu, n)){
    printf("OK\n");
  //Fct test egale
printf("Test egale\n");
    if(is_equal(T_cpu, T_gpu, n))
      printf("OK\n");
    else
      printf("[error] T_gpu mal trie");
  }else{
    printf("[error] T_cpu mal trie");   
  }

//Time resuts
printf("\nTime results\n");
  float millisecondsGPU = 0;
  hipEventElapsedTime(&millisecondsGPU, startGPU, stopGPU);
  printf("\nTime GPU : %f ms\n",millisecondsGPU);
  float millisecondsCPU = 0;
  hipEventElapsedTime(&millisecondsCPU, startCPU, stopCPU);
  printf("Time CPU : %f ms\n",millisecondsCPU);

printf("\n");
  char s;
  printf("Appuyer sur ENTREE pour continuer\n");
  s=getchar();
  putchar(s);

//free
  free(T_cpu);
  free(T_gpu);
}
//4 Exemple d'application donnee automobile
void tabCars(){
  std::ifstream file;
  int x;
  int Entree = 0;
  int Erreur = 0;
  while(Entree == 0){
    system("clear");
    std::cout<<"Exemple d'application données automobiles"<<std::endl;
    std::cout<<"0\tTop 10 Prix"<<std::endl;
    std::cout<<"1\tTop 10 Km"<<std::endl;
    if(Erreur == 1){
      std::cout<<"[Erreur] Entrée non valide ! choisir une option 1 ou 2"<<std::endl;
      Erreur  =  0;
    }
    if(Erreur == 2){
      std::cout<<"[Erreur] Fichier price.txt manquant! Lancer le script doc.py"<<std::endl;
      Erreur  =  0;
    }
    if(Erreur == 3){
      std::cout<<"[Erreur] Fichier kilometer.txt manquant! Lancer le script doc.py"<<std::endl;
      Erreur  =  0;
    }
    std::cout<<"Entrer votre choix (0 ou 1) puis valider avec la touche ENTREE"<<std::endl;
    std::cin>>x;
    while (getchar() != '\n'); //vide le buffer de saisie
    if(x==0){
      file.open("../data/price.txt");//ouverture fichier
      if(file.is_open()){
        char s;
        std::cout<<"Vous avez entré "<<x<<std::endl;
        printf("Appuyer sur ENTREE pour continuer\n");
        s=getchar();
        putchar(s);
        Entree = 1;
      }else{
        Erreur  =  2;
      }
    }else{
      if(x==1){
        file.open("../data/kilometer.txt");//ouverture fichier
        if(file.is_open()){
          char s;
          std::cout<<"Vous avez entré "<<x<<std::endl;
          printf("Appuyer sur ENTREE pour continuer\n");
          s=getchar();
          putchar(s);
          Entree = 2;
        }else{
          Erreur  =  3;
        }
      }else{
        Erreur  =  1; //mauvais choix
      }
    }
  }


//lecture fichier
printf("Lecture fichier...\n");
  int num;
  std::vector<int> tab;
  while (file >> num) { tab.push_back(num); }

  int n = tab.size();
printf("%d elements lus\n",n);

// var pour timer
  hipEvent_t startCPU, stopCPU;
  hipEventCreate ( &startCPU );
  hipEventCreate ( &stopCPU );
  hipEvent_t startGPU, stopGPU;
  hipEventCreate ( &startGPU );
  hipEventCreate ( &stopGPU );

//Alloc Array
printf("Alloc Array\n");
  srand(time(NULL));
  int* T_cpu = (int*)malloc(n*sizeof(int));
  int* T_gpu = (int*)malloc(n*sizeof(int));
  int cpt=0;

//init Array
printf("\nInit Array\n");
  while (cpt<n){
    T_gpu[cpt]= tab[cpt];
    T_cpu[cpt] = tab[cpt];
    cpt++;
  }
printf("\n");

//sort CPU
printf("\nCall sort CPU\n");
  hipEventRecord(startCPU);
  mergeAndSortRecuGPU(T_gpu, 0, n-1);
  hipEventRecord(stopCPU);
printf("\n");
//sort GPU
printf("\nCall sort GPU\n");
  hipEventRecord(startGPU);
  mergeAndSortRecuCPU(T_cpu, n);
  hipEventRecord(stopGPU);
printf("Call cudaDeviceSynchronize\n");
  hipDeviceSynchronize();

//test tri ok?
printf("\nTest tri\n");
  if(is_sorted(T_cpu, n)){
    printf("OK\n");
  //Fct test egale
printf("Test egale\n");
    if(is_equal(T_cpu, T_gpu, n))
      printf("OK\n");
    else
      printf("[error] T_gpu mal trie");
  }else{
    printf("[error] T_cpu mal trie");   
  }

//Time resuts
printf("\nTime results\n");
  float millisecondsGPU = 0;
  hipEventElapsedTime(&millisecondsGPU, startGPU, stopGPU);
  printf("\nTime GPU : %f ms\n",millisecondsGPU);
  float millisecondsCPU = 0;
  hipEventElapsedTime(&millisecondsCPU, startCPU, stopCPU);
  printf("Time CPU : %f ms\n",millisecondsCPU);
printf("\n");
  if(Entree == 1){//price
  printf("\nTop 10 des prix les plus élevés des voitures:\n");
    cpt = n;
    while (cpt>n-10){
      printf("[%d] %d Euros\n",n-cpt+1,T_gpu[cpt-1]);
      cpt--;
    }
   printf("\n");
  }
  if(Entree == 2){//kilometer
  printf("\nTop 10 des voitures les plus durables :\n");
    cpt = n;
    while (cpt>n-10){
      printf("[%d] %d Km\n",n-cpt+1,T_gpu[cpt-1]);
      cpt--;
    }
   printf("\n");
  }

printf("\n");
  char s;
  printf("Appuyer sur ENTREE pour continuer\n");
  s=getchar();
  putchar(s);

//free
  free(T_cpu);
  free(T_gpu);
}

int sousMenu(){
  int x;
  int Entree = 0;
  int Erreur = 0;
  while(Entree == 0){
    system("clear");
    std::cout<<"Tester l'algorithme de tri"<<std::endl;
    std::cout<<"0\tQuitter"<<std::endl;
    std::cout<<"1\tHelp"<<std::endl;
    std::cout<<"2\tTest sur tableau random"<<std::endl;
    std::cout<<"3\tLecture à partir d'un fichier"<<std::endl;
    std::cout<<"4\tExemple d'application données automobiles"<<std::endl;
    std::cout<<"5\tRetour menu principal"<<std::endl;

    if(Erreur == 1){
      std::cout<<"[Erreur] Entrée non valide !"<<std::endl;
      Erreur  =  0;
    }
    std::cout<<"Appuyer sur une touche entre 0 et 5  puis valider avec la touche ENTREE"<<std::endl;
    std::cin>>x;
    while (getchar() != '\n'); //vide le buffer de saisie

    switch ( x ){
      case 0: 
        system("clear");
        std::cout<<"Vous avez choisi de quitter"<<std::endl;
        std::cout<<"Etes vous sûr?"<<std::endl;
        std::cout<<"Appuyer sur 1 pour revenir au Menu principal sinon une touche pour quitter puis valider avec la touche ENTREE"<<std::endl;
        std::cin>>x;
        while (getchar() != '\n'); //vide le buffer de saisie
        if(x!=1){
          system("clear");
          std::cout<<"Merci et à bientot"<<std::endl;
          sleep(0.6);
          system("clear");
          std::cout<<"Merci et à bientot"<<std::endl;
          std::cout<<"3"<<std::endl;
           sleep(1);
          system("clear");
          std::cout<<"Merci et à bientot"<<std::endl;
          std::cout<<"  2"<<std::endl;
          sleep(1);
          system("clear");
          std::cout<<"Merci et à bientot"<<std::endl;
          std::cout<<"    1"<<std::endl;
          sleep(1);
          Entree = 2;
          system("clear");
        }
        break; 
      case 1:  
        system("clear");
        std::cout<<"1\tHelp"<<std::endl;
        std::cout<<"Merci d'utiliser exclusivement les CHIFFRES pour les selections dans les menus"<<std::endl;
        std::cout<<""<<std::endl;
        std::cout<<""<<std::endl;
        std::cout<<"Naviguer dans les menus pour tester toutes nos implementations de tri merge and sort sur CPU et GPU."<<std::endl;
        std::cout<<""<<std::endl;
        std::cout<<"Utilisation :"<<std::endl;
        std::cout<<"- Test sur tableau random : génère aléatoirement un tableau d'entier et le tri"<<std::endl;
        std::cout<<"- Lecture à partir d'un fichier : lit à partir d'un fichier un tableau d'entier et le tri."<<std::endl;
        std::cout<<"  Le fichier doit être écrit de la meme manière que data/exemple.txt"<<std::endl;
        std::cout<<"  On peut utiliser script/genere.c pour en créer un nouveau."<<std::endl;
std::cout<<"- Exemple d'application données automobiles : des données automobiles sont lues et triees."<<std::endl;
std::cout<<"  Il est essentiel de lancer le script de pré-traitrement des données script/doc.py avant."<<std::endl;
        std::cout<<""<<std::endl;
        std::cout<<""<<std::endl;
        std::cout<<""<<std::endl;
        sleep(2);
        char sh;
        printf("Appuyer sur ENTREE pour continuer\n");
        sh=getchar();
        putchar(sh);
        break;  
      case 2:  
        // Test sur tableau random
        tabRandom();
        break;
      case 3:  
      // Lecture à partir d'un fichier
        tabFile();
        break;
      case 4:  
      // Exemple d'application donnee automobile
        tabCars();
        break;
      case 5:  
        Entree = 1;
        break;
      default:  
        Erreur = 1;
    } 
  }
  return Entree;
}

void menuPrincipal(){
  int x;
  int Entree = 0;
  int Erreur = 0;
  while(Entree == 0){
    system("clear");
    std::cout<<"Menu principal"<<std::endl;
    std::cout<<"0\tQuitter"<<std::endl;
    std::cout<<"1\tHelp"<<std::endl;
    std::cout<<"2\tTester l'algorithme de tri"<<std::endl;
    std::cout<<"3\tReset"<<std::endl;

    if(Erreur == 1){
      std::cout<<"[Erreur] Entrée non valide !"<<std::endl;
      Erreur  =  0;
    }
    std::cout<<"Appuyer sur une touche entre 0 et 3  puis valider avec la touche ENTREE"<<std::endl;
    std::cin>>x;
    while (getchar() != '\n'); //vide le buffer de saisie
    switch ( x ){
      case 0: 
        system("clear");
        std::cout<<"Vous avez choisi de quitter"<<std::endl;
        std::cout<<"Etes vous sûr?"<<std::endl;
        std::cout<<"Appuyer sur 1 pour revenir au Menu principal sinon une touche pour quitter puis valider avec la touche ENTREE"<<std::endl;
        std::cin>>x;
        while (getchar() != '\n'); //vide le buffer de saisie
        if(x!=1){
          system("clear");
          std::cout<<"Merci et à bientôt"<<std::endl;
          sleep(0.6);
          system("clear");
          std::cout<<"Merci et à bientôt"<<std::endl;
          std::cout<<"3"<<std::endl;
          sleep(1);
          system("clear");
          std::cout<<"Merci et à bientôt"<<std::endl;
          std::cout<<"  2"<<std::endl;
          sleep(1);
          system("clear");
          std::cout<<"Merci et à bientôt"<<std::endl;
          std::cout<<"    1"<<std::endl;
          sleep(1);
          Entree = 1;
          system("clear");
        }
        break; 
      case 1:  
        system("clear");
        std::cout<<"1\tHelp"<<std::endl;
        std::cout<<"Merci d'utiliser exclusivement les CHIFFRES pour les sélections dans les menus"<<std::endl;
        std::cout<<""<<std::endl;
        std::cout<<""<<std::endl;
        std::cout<<"Naviguer dans les menus pour tester toutes nos implémentations de tri merge and sort sur CPU et GPU."<<std::endl;
        std::cout<<""<<std::endl;
        sleep(2);
        char sh;
        printf("Appuyer sur ENTREE pour continuer\n");
        sh=getchar();
        putchar(sh);
        break;  
      case 2:  
        if(sousMenu()==2)
          Entree = 1;
        break;
      case 3:  
        intro();
        break;
      default:  
        Erreur = 1;
    } 
  }
}

// End
void end(){
  system("clear");
  printf("\nBy\n");
  printf("ALOUI Driss\n");
  printf("DO Alexandre\n");
  printf("\nNovembre 2017\n");
}

void prelude(){
  std::cout<<"Naviguer dans les menus pour tester toutes nos implémentations de tri merge and sort sur CPU et GPU."<<std::endl;
  std::cout<<""<<std::endl;
  sleep(2);
  char sh;
  printf("Appuyer sur ENTREE pour continuer\n");
  sh=getchar();
  putchar(sh);
}

//---------------- MAIN ----------------
//main
int main(){
  //int n = TAILLE;

// Gestion de l intro
  FILE* fichier = NULL;
  fichier = fopen("tmp.txt", "r");

  if (fichier == NULL){ // 1ere fois
    intro();
    prelude();
    FILE* fichier2 = NULL;
    fichier2 = fopen("tmp.txt", "w");
    fprintf(fichier2, "1");
    fclose(fichier2);
  }else{ // intro deja faite une fois)
    fclose(fichier);
    introShort();
  }
  
  menuPrincipal();
  end();

  return 0;
}
